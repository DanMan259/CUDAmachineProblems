#include "hip/hip_runtime.h"
/*
	Name: Daniyal Manair
	Student Number: 20064993
*/
#include "hip/hip_runtime.h"
#include ""

#include <vector>
#include <stdio.h>
#include <random>
#include <algorithm>
#include <chrono>
#include <map>

__global__ void TiledMatrixMulGPU(float* A, float* B, float* C, const int M, const int N, const int K) {
	__shared__ float t_A [TILE_WIDTH][TILE_WIDTH];
	__shared__ float t_B [TILE_WIDTH][TILE_WIDTH];
	
	unsigned int bx = blockIdx.x;
	unsigned int by = blockIdx.x;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;
	unsigned int row = by * blockDim.y + ty;
	unsigned int col = bx * blockDim.x + tx;
	
	float cValue = 0.0;
	for (int i = 0; i < (N/TILE_WIDTH); i++){
		t_A[ty][tx] = A[row*N + i*TILE_WIDTH+tx];
		t_B[ty][tx] = B[(i*TILE_WIDTH+ty)*K + col];	
		__syncthreads();
		for(int j = 0; j < (TILE_WIDTH); j++)
			cValue += t_A[ty][j] *t_B[j][tx];
		__syncthreads();
	}
	C[row*K+col] = cValue;
}

void initialData(float* matrix, const int N){
	for (int i = 0; i < (N*N); i++)
		matrix[i] = (float)(rand() & 0xFF) / 10.0f;
}

void MatrixMulCPU(float* A, float* B, float* C, const int N){
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			for (int k = 0; k < N; k++) 
				C[i * N + j] += A[i * N + k] * B[k * N + j];
		}
	}
}

void checkResult(float* CPU, float* GPU, const int N) {
	double epsilon = 1.0E-8;
	
	for (int i = 0; i < (N*N); i++){
		if (abs(CPU[i] - GPU[i]) > epsilon){
			printf("CPU %f GPU %f ", CPU[i], GPU[i]);
			printf("Arrays do not match.\n\n");
			return;
		}
	}
	printf("Test PASSED\n\n");
}

void printArr(float* matrix, const int N) {
	printf("[");
	for (int i = 0; i < (N*N); i++)
		printf("%f,", matrix[i]);
	printf("\b]\n");

}

float GPUtest(float* C_A, float* C_B, float* CPUResult, const int blockSize, const int N){
	// Initialize variables
	hipEvent_t gStart, gEnd;
	float timeDuration;
	float *G_A, *G_B, *G_C, *GPUResult;
	size_t size = N * N * sizeof(float);
	
	// Initialize GPU variables
	hipMalloc((void**)&G_A, size);
	hipMalloc((void**)&G_B, size);
	hipMalloc((void**)&G_C, size);
	GPUResult = (float*)malloc(size);
	memset(GPUResult, 0.0, size);
	hipEventCreate(&gStart);
    hipEventCreate(&gEnd);
	
	// Copy over the data
	hipMemcpy(G_A, C_A, size, hipMemcpyHostToDevice);
	hipMemcpy(G_B, C_B, size, hipMemcpyHostToDevice);
	
	// Perform GPU comparison
	if (blockSize == 0){
		hipEventRecord(gStart);
		MatrixMulGPUSingle <<<1, 1>>> (G_A, G_B, G_C, N);
		hipEventRecord(gEnd);
	} else {
		// Create block
		int numBlocks = N / blockSize;
		if (N % blockSize) numBlocks++;
		dim3 block(blockSize, blockSize, 1);
		dim3 grid(numBlocks, numBlocks, 1);

		hipEventRecord(gStart);
		MatrixMulGPU <<<grid, block >>> (G_A, G_B, G_C, N);
		hipEventRecord(gEnd);
	}
		
	
	hipEventSynchronize(gEnd);
	hipEventElapsedTime(&timeDuration, gStart, gEnd);
	
	hipMemcpy(GPUResult, G_C, size, hipMemcpyDeviceToHost);
	checkResult(CPUResult, GPUResult, N);
	
	hipFree(G_A);
	hipFree(G_B);
	hipFree(G_C);
	free(GPUResult);
	return timeDuration;
}

void computeMatrix(const int N) {
	// Initial prints
	printf("------------------------------------------------------------------------\n\n");
	printf("%dx%d matrix multiplication.\n\n", N, N);

	// Initialize Host variables
	float *C_A, *C_B, *C_C;
	size_t size = N * N * sizeof(float);
	FILE *fp;
	
	// Initialize space
	C_A = (float*)malloc(size);
	C_B = (float*)malloc(size);
	C_C = (float*)malloc(size);
	fp=fopen("machineProblem3.csv","a");
	
	// Set with random data
	initialData(C_A, N);
	initialData(C_B, N);
	memset(C_C, 0.0, size);

	// Serial Test CPU
	auto cStart = std::chrono::high_resolution_clock::now();
	MatrixMulCPU(C_A, C_B, C_C, N);
	auto cEnd = std::chrono::high_resolution_clock::now();
	auto timeElapse = (std::chrono::duration_cast<std::chrono::microseconds>(cEnd - cStart).count())/1000.0;
	printf("The CPU took %f to perform the computation.\n\n", timeElapse);
	fprintf(fp,"%d,CPU,0,%f\n",N,timeElapse);
	
	// Test Complete parallel Computation
	int blockSizes [] = {0, 2, 4, 10, 20, 25};
	float timeDuration;
	
	for (int i = 0; i < 6; i++){
		timeDuration = GPUtest(C_A, C_B, C_C, blockSizes[i], N);
		printf("The GPU took %f to perform the computation with block size %d.\n", timeDuration, blockSizes[i]);
		fprintf(fp,"%d,GPU,%d,%f\n",N,blockSizes[i],timeDuration);
	}
	
	// Free all the memory
	free(C_A);
	free(C_B);
	free(C_C);
	fclose(fp);
	hipDeviceReset();
}

int main(){
	FILE *fp;
	fp=fopen("machineProblem3.csv","w");
	fprintf(fp,"matrixSize,processor,blockSize,time\n");
	fclose(fp);
	int matrixWidths [] = {100, 200, 500, 1000, 1500, 5000};
	
	for (int i = 0; i < 6; i++)
		computeMatrix(matrixWidths[i]);

	printf("------------------------------------------------------------------------\n\n");
    
	return 0;
}
