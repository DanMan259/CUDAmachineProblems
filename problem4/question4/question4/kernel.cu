#include "hip/hip_runtime.h"
/*
	Name: Daniyal Manair
	Student Number: 20064993
*/
#include "hip/hip_runtime.h"
#include ""

#include <vector>
#include <stdio.h>
#include <random>
#include <algorithm>
#include <chrono>
#include <map>

__global__ void TiledMatrixMulGPU2(float* A, float* B, float* C, const int N) {
	__shared__ float t_A [2][2];
	__shared__ float t_B [2][2];

	unsigned int tileWidth = 2;
	unsigned int bx = blockIdx.x;
	unsigned int by = blockIdx.y;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;
	unsigned int row = by * tileWidth + ty;
	unsigned int col = bx * tileWidth + tx;

	float cValue = 0.0;
	for (int i = 0; i < (N / tileWidth); i++) {
		t_A[ty][tx] = A[row*N + i*tileWidth + tx];
		t_B[ty][tx] = B[(i*tileWidth + ty)*N + col];
		__syncthreads();
		for (int j = 0; j < tileWidth; j++)
			cValue += t_A[ty][j] * t_B[j][tx];
		__syncthreads();
	}
	C[row*N + col] = cValue;
}

__global__ void TiledMatrixMulGPU4(float* A, float* B, float* C, const int N) {
	__shared__ float t_A [4][4];
	__shared__ float t_B [4][4];

	unsigned int tileWidth = 4;
	unsigned int bx = blockIdx.x;
	unsigned int by = blockIdx.y;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;
	unsigned int row = by * tileWidth + ty;
	unsigned int col = bx * tileWidth + tx;

	float cValue = 0.0;
	for (int i = 0; i < (N / tileWidth); i++) {
		t_A[ty][tx] = A[row*N + i*tileWidth + tx];
		t_B[ty][tx] = B[(i*tileWidth + ty)*N + col];
		__syncthreads();
		for (int j = 0; j < tileWidth; j++)
			cValue += t_A[ty][j] * t_B[j][tx];
		__syncthreads();
	}
	C[row*N + col] = cValue;
}	


__global__ void TiledMatrixMulGPU10(float* A, float* B, float* C, const int N) {
	__shared__ float t_A [10][10];
	__shared__ float t_B [10][10];

	unsigned int tileWidth = 10;
	unsigned int bx = blockIdx.x;
	unsigned int by = blockIdx.y;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;
	unsigned int row = by * tileWidth + ty;
	unsigned int col = bx * tileWidth + tx;

	float cValue = 0.0;
	for (int i = 0; i < (N / tileWidth); i++) {
		t_A[ty][tx] = A[row*N + i*tileWidth + tx];
		t_B[ty][tx] = B[(i*tileWidth + ty)*N + col];
		__syncthreads();
		for (int j = 0; j < tileWidth; j++)
			cValue += t_A[ty][j] * t_B[j][tx];
		__syncthreads();
	}
	C[row*N + col] = cValue;
}

__global__ void TiledMatrixMulGPU20(float* A, float* B, float* C, const int N) {
	__shared__ float t_A [20][20];
	__shared__ float t_B [20][20];

	unsigned int tileWidth = 20;
	unsigned int bx = blockIdx.x;
	unsigned int by = blockIdx.y;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;
	unsigned int row = by * tileWidth + ty;
	unsigned int col = bx * tileWidth + tx;

	float cValue = 0.0;
	for (int i = 0; i < (N / tileWidth); i++) {
		t_A[ty][tx] = A[row*N + i*tileWidth + tx];
		t_B[ty][tx] = B[(i*tileWidth + ty)*N + col];
		__syncthreads();
		for (int j = 0; j < tileWidth; j++)
			cValue += t_A[ty][j] * t_B[j][tx];
		__syncthreads();
	}
	C[row*N + col] = cValue;
}

__global__ void TiledMatrixMulGPU25(float* A, float* B, float* C, const int N, const int tileWidth) {
	__shared__ float t_A [25][25];
	__shared__ float t_B [25][25];

	unsigned int tileWidth = 25;
	unsigned int bx = blockIdx.x;
	unsigned int by = blockIdx.y;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;
	unsigned int row = by * tileWidth + ty;
	unsigned int col = bx * tileWidth + tx;

	float cValue = 0.0;
	for (int i = 0; i < (N / tileWidth); i++) {
		t_A[ty][tx] = A[row*N + i*tileWidth + tx];
		t_B[ty][tx] = B[(i*tileWidth + ty)*N + col];
		__syncthreads();
		for (int j = 0; j < tileWidth; j++)
			cValue += t_A[ty][j] * t_B[j][tx];
		__syncthreads();
	}
	C[row*N + col] = cValue;
}

void initialData(float* matrix, const int size){
	for (int i = 0; i < size; i++)
		matrix[i] = (float)(rand() & 0xFF) / 10.0f;
}

void MatrixMulCPU(float* A, float* B, float* C, const int N){
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			for (int k = 0; k < N; k++) 
				C[i * N + j] += A[i * N + k] * B[k * N + j];
		}
	}
}

void checkResult(float* CPU, float* GPU, const int size) {
	double epsilon = 1.0E-8;
	
	for (int i = 0; i < size; i++){
		if (abs(CPU[i] - GPU[i]) > epsilon){
			printf("CPU %f GPU %f ", CPU[i], GPU[i]);
			printf("Arrays do not match.\n\n");
			return;
		}
	}
	printf("Test PASSED\n\n");
}

void printArr(float* matrix, const int size) {
	printf("[");
	for (int i = 0; i < size; i++)
		printf("%f,", matrix[i]);
	printf("\b]\n");

}

void GPUtest(float* C_A, float* C_B, float* CPUResult, const int tileSize, const int N){
	// Initialize variables
	hipEvent_t gStart, gEnd;
	float timeDuration;
	float *G_A, *G_B, *G_C, *GPUResult;
	size_t size = N * N * sizeof(float);
	
	// Initialize GPU variables
	hipMalloc((void**)&G_A, size);
	hipMalloc((void**)&G_B, size);
	hipMalloc((void**)&G_C, size);
	GPUResult = (float*)malloc(size);
	memset(GPUResult, 0.0, size);
	hipEventCreate(&gStart);
    hipEventCreate(&gEnd);
	
	// Copy over the data
	hipMemcpy(G_A, C_A, size, hipMemcpyHostToDevice);
	hipMemcpy(G_B, C_B, size, hipMemcpyHostToDevice);
	
	// Perform GPU comparison

	// Create block
	dim3 block(tileSize, tileSize, 1);
	dim3 grid(N/tileSize, N/tileSize, 1);

	hipEventRecord(gStart);
	if (tileSize == 2)
		TiledMatrixMulGPU2 <<<grid, block>>> (G_A, G_B, G_C, N, tileSize);
	else if (tileSize == 4)
		TiledMatrixMulGPU4 <<<grid, block>>> (G_A, G_B, G_C, N, tileSize);
	else if (tileSize == 10)
		TiledMatrixMulGPU10 <<<grid, block>>> (G_A, G_B, G_C, N, tileSize);
	else if (tileSize == 20)
		TiledMatrixMulGPU20 <<<grid, block>>> (G_A, G_B, G_C, N, tileSize);
	else if (tileSize == 25)
		TiledMatrixMulGPU25 <<<grid, block>>> (G_A, G_B, G_C, N, tileSize);
	hipEventRecord(gEnd);
	
	hipEventSynchronize(gEnd);
	hipEventElapsedTime(&timeDuration, gStart, gEnd);
	
	printArr(GPUResult, N*N);
	hipMemcpy(GPUResult, G_C, size, hipMemcpyDeviceToHost);
	checkResult(CPUResult, GPUResult, N*N);
	
	hipFree(G_A);
	hipFree(G_B);
	hipFree(G_C);
	free(GPUResult);
	
	FILE *fp;
	fp=fopen("machineProblem4.csv","a");
	printf("The GPU took %f to perform the computation with tile size %d.\n", timeDuration, tileSize);
	fprintf(fp,"%d,%d,%f\n",N,tileSize,timeDuration);
	fclose(fp);
}

void computeMatrix(const int N) {
	// Initial prints
	printf("------------------------------------------------------------------------\n\n");
	printf("%dx%d matrix multiplication.\n\n", N, N);

	// Initialize Host variables
	float *C_A, *C_B, *C_C;
	size_t size = N * N * sizeof(float);
	
	// Initialize space
	C_A = (float*)malloc(size);
	C_B = (float*)malloc(size);
	C_C = (float*)malloc(size);
	
	// Set with random data
	initialData(C_A, N*N);
	initialData(C_B, N*N);
	memset(C_C, 0.0, size);

	// Serial Test CPU
	MatrixMulCPU(C_A, C_B, C_C, N);

	printArr(C_A, N*N);
	printArr(C_B, N*N);
	printArr(C_C, N*N);
	
	// Test Complete parallel Computation
	int tileSizes [] = {2, 4, 10, 20, 25};
	
	for (int i = 4; i < 5; i++)
		GPUtest(C_A, C_B, C_C, tileSizes[i], N);
	
	// Free all the memory
	free(C_A);
	free(C_B);
	free(C_C);
	hipDeviceReset();
}

// --------------------BONUS-----------------------------

#define BONUSTILE_1C 8
#define BONUSTILE_1R 8
#define BONUSTILE_2C 14
#define BONUSTILE_2R 14

__global__ void TiledMatrixMulGPUBonus(float* A, float* B, float* C, const int M, const int N, const int K, const int tileCase) {
	
	if (tileCase == 1){
		__shared__ float t_A [BONUSTILE_1R][BONUSTILE_1C];
		__shared__ float t_B [BONUSTILE_1R][BONUSTILE_1C];

		unsigned int bx = blockIdx.x;
		unsigned int by = blockIdx.x;
		unsigned int tx = threadIdx.x;
		unsigned int ty = threadIdx.y;
		unsigned int row = by * blockDim.y + ty;
		unsigned int col = bx * blockDim.x + tx;

		float cValue = 0.0;
		for (int i = 0; i < ((BONUSTILE_1C + N - 1) / BONUSTILE_1C); i++) {
			if (i*BONUSTILE_1C + tx < N && row < M) {
				t_A[ty][tx] = A[row*N + i*BONUSTILE_1C + tx];
			}
			else {
				t_A[ty][tx] = 0;
			}

			if ((i*BONUSTILE_1C + ty < N) && col < K) {
				t_B[ty][tx] = B[(i*BONUSTILE_1C + ty)*K + col];
			}
			else {
				t_B[ty][tx] = 0;
			}
			__syncthreads();
			for (int j = 0; j < BONUSTILE_1C; j++)
				cValue += t_A[ty][j] * t_B[j][tx];
			__syncthreads();
		}

		if (row < M && col < K)
			C[row*K + col] = cValue;
	} else {
		__shared__ float t_A [BONUSTILE_2R][BONUSTILE_2C];
		__shared__ float t_B [BONUSTILE_2R][BONUSTILE_2C];

		unsigned int bx = blockIdx.x;
		unsigned int by = blockIdx.x;
		unsigned int tx = threadIdx.x;
		unsigned int ty = threadIdx.y;
		unsigned int row = by * blockDim.y + ty;
		unsigned int col = bx * blockDim.x + tx;

		float cValue = 0.0;
		for (int i = 0; i < ((BONUSTILE_2C + N - 1) / BONUSTILE_2C); i++) {
			if (i*BONUSTILE_2C + tx < N && row < M) {
				t_A[ty][tx] = A[row*N + i*BONUSTILE_2C + tx];
			}
			else {
				t_A[ty][tx] = 0;
			}

			if ((i*BONUSTILE_2C + ty < N) && col < K) {
				t_B[ty][tx] = B[(i*BONUSTILE_2C + ty)*K + col];
			}
			else {
				t_B[ty][tx] = 0;
			}
			__syncthreads();
			for (int j = 0; j < BONUSTILE_2C; j++)
				cValue += t_A[ty][j] * t_B[j][tx];
			__syncthreads();
		}

		if (row < M && col < K)
			C[row*K + col] = cValue;
	
	}
}

void computeMatrixBonus(const int M, const int N, const int K) {
	// Initial prints
	printf("------------------------------------------------------------------------\n\n");
	printf("%dx%d and %dx%d matrix multiplication.\n\n", M, N, N, K);

	float *C_A, *C_B, *C_C, *GPUResult;
	size_t sizeA = M * N * sizeof(float);
	size_t sizeB = N * K * sizeof(float);
	size_t sizeC = M * K * sizeof(float);
	
	// Initialize space
	C_A = (float*)malloc(sizeA);
	C_B = (float*)malloc(sizeB);
	C_C = (float*)malloc(sizeC);
	GPUResult = (float*)malloc(sizeC);
	
	// Set with random data
	initialData(C_A, M * N);
	initialData(C_B, N * K);
	memset(C_C, 0.0, sizeC);
	memset(GPUResult, 0.0, sizeC);

	// Serial Test CPU
	MatrixMulCPU(C_A, C_B, C_C, N);
	
	// GPU calculations
	float *G_A, *G_B, *G_C;
	
	// Initialize GPU variables
	hipMalloc((void**)&G_A, sizeA);
	hipMalloc((void**)&G_B, sizeB);
	hipMalloc((void**)&G_C, sizeC);
	// Copy over values
	hipMemcpy(G_A, C_A, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(G_B, C_B, sizeB, hipMemcpyHostToDevice);
	
	dim3 block(8, 8);
	dim3 thread((int)ceil((K + block.x - 1)) / block.x, (int)ceil((M + block.y - 1) / block.y));
	
	// Test different tile sizes
	
	// Case 1 8x8
	dim3 block1(BONUSTILE_1R, BONUSTILE_1C);
	dim3 grid1((int)ceil((K + block.x - 1)) / block.x, (int)ceil((M + block.y - 1) / block.y));
	
	TiledMatrixMulGPUBonus <<<grid1, block1>>> (G_A, G_B, G_C, M, N, K, 1);
	hipMemcpy(GPUResult, G_C, sizeC, hipMemcpyDeviceToHost);
	checkResult(C_C, GPUResult, N*N);
	
	// Case 2 14x14
	dim3 block2(BONUSTILE_2R, BONUSTILE_2C);
	dim3 grid2((int)ceil((K + block.x - 1)) / block.x, (int)ceil((M + block.y - 1) / block.y));
	
	TiledMatrixMulGPUBonus <<<grid2, block2>>> (G_A, G_B, G_C, M, N, K, 2);
	hipMemcpy(GPUResult, G_C, sizeC, hipMemcpyDeviceToHost);
	checkResult(C_C, GPUResult, N*N);
	

	// Free all the memory
	free(C_A);
	free(C_B);
	free(C_C);
	free(GPUResult);
	hipFree(G_A);
	hipFree(G_B);
	hipFree(G_C);
	hipDeviceReset();
}

// ------------------------------------------------------


int main(){
	FILE *fp;
	fp=fopen("machineProblem4.csv","w");
	fprintf(fp,"matrixSize,tileSize,time\n");
	fclose(fp);
	int matrixWidths [] = {2, 100, 200, 500, 1000, 1500, 5000};
	
	for (int i = 0; i < 1; i++)
		computeMatrix(matrixWidths[i]);

	printf("------------------------------------------------------------------------\n\n");
    
	printf("BONUS\n");
	
	//computeMatrixBonus(250, 300, 450);
	
	return 0;
}
