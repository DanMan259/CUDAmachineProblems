#include "hip/hip_runtime.h"
/*
	Name: Daniyal Manair
	Student Number: 20064993
*/
#include "hip/hip_runtime.h"
#include ""

#include <vector>
#include <stdio.h>
#include <random>
#include <algorithm>
#include <chrono>
#include <map>

__global__ void sumMatrixGPU(float* A, float* B, float* C, const int N) {
	unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int idx = row * N + col;
	if (row < N && col < N){
		C[idx] = A[idx] + B[idx];
	}
}

__global__ void sumMatrixGPUperRow(float* A, float* B, float* C, const int N) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < N){
		for (int i = 0; i < N; i++)
			C[row * N + i] = A[row * N + i] + B[row * N + i];
	}
}

__global__ void sumMatrixGPUperCol(float* A, float* B, float* C, const int N) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (col < N){
		for (int i = 0; i < N; i++)
			C[i * N + col] = A[i * N + col] + B[i * N + col];
	}
}

void initialData(float* matrix, const int N){
	for (int i = 0; i < (N*N); i++)
		matrix[i] = (float)(rand() & 0xFF) / 10.0f;
}

void sumMatrixCPU(float* A, float* B, float* C, const int N){
	for (int i = 0; i < (N*N); i++)
		C[i] = A[i] + B[i];
}

void checkResult(float* CPU, float* GPU, const int N) {
	double epsilon = 1.0E-8;
	
	for (int i = 0; i < (N*N); i++){
		if (abs(CPU[i] - GPU[i]) > epsilon){
			printf("CPU %f GPU %f ", CPU[i], GPU[i]);
			printf("Arrays do not match.\n\n");
			return;
		}
	}
	printf("Test PASSED\n\n");
}

void printArr(float* matrix, const int N) {
	printf("[");
	for (int i = 0; i < (N*N); i++)
		printf("%f,", matrix[i]);
	printf("\b]\n");

}

void computeMatrix(const int N) {
	// Initial prints
	printf("------------------------------------------------------------------------\n\n");
	printf("%dx%d matrix addition.\n\n", N, N);

	// Initialize Host variables
	float* C_A, *C_B, *C_C, *C_C1;
	size_t size = N * N * sizeof(float);
	
	// Initialize space
	C_A = (float*)malloc(size);
	C_B = (float*)malloc(size);
	C_C = (float*)malloc(size);
	C_C1 = (float*)malloc(size);

	// Set with random data
	initialData(C_A, N);
	initialData(C_B, N);
	memset(C_C, 0, N);
	memset(C_C1, 0, N);

	// Initialize GPU variables
	float* G_A, *G_B, *G_C, *G_C1, *G_C2;
	hipMalloc((void**)&G_A, size);
	hipMalloc((void**)&G_B, size);
	hipMalloc((void**)&G_C, size);
	hipMalloc((void**)&G_C1, size);
	hipMalloc((void**)&G_C2, size);
	
	// Copy over the data
	hipMemcpy(G_A, C_A, size, hipMemcpyHostToDevice);
	hipMemcpy(G_B, C_B, size, hipMemcpyHostToDevice);

	// Serial Test CPU
	auto start = std::chrono::high_resolution_clock::now();
	sumMatrixCPU(C_A, C_B, C_C, N);
	auto end = std::chrono::high_resolution_clock::now();
	auto timeElapse = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	printf("The CPU took %d to perform the computation.\n\n", timeElapse);
	
	
	// Test Complete parallel Computation
	dim3 block(16, 16);
	dim3 thread((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);
	
	start = std::chrono::high_resolution_clock::now();
	hipDeviceSynchronize();
	sumMatrixGPU <<<thread, block >>> (G_A, G_B, G_C, N);
	hipDeviceSynchronize();
	end = std::chrono::high_resolution_clock::now();
	timeElapse = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	printf("The GPU took %d to perform the computation with one thread per element.\n", timeElapse);
	
	// Copy over the result and compare
	hipMemcpy(C_C1, G_C, size, hipMemcpyDeviceToHost);
	checkResult(C_C, C_C1, N);
	
	// Test row based parallel Computation
	dim3 block(16);
	dim3 thread((N + block.x - 1) / block.x);
	
	start = std::chrono::high_resolution_clock::now();
	hipDeviceSynchronize();
	sumMatrixGPUperRow <<<thread1, block1 >>> (G_A, G_B, G_C1, N);
	hipDeviceSynchronize();
	end = std::chrono::high_resolution_clock::now();
	timeElapse = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	printf("The GPU took %d to perform the computation with one thread per Row.\n", timeElapse);

	// Copy over the result and compare
	hipMemcpy(C_C1, G_C1, size, hipMemcpyDeviceToHost);
	checkResult(C_C, C_C1, N);

	// Test Complete parallel Computation
	dim3 block(16);
	dim3 thread((N + block.x - 1) / block.x);
	
	// Test column based parallel Computation
	start = std::chrono::high_resolution_clock::now();
	hipDeviceSynchronize();
	sumMatrixGPUperCol <<<thread2, block2 >>> (G_A, G_B, G_C2, N);
	hipDeviceSynchronize();
	end = std::chrono::high_resolution_clock::now();
	timeElapse = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	printf("The GPU took %d to perform the computation with one thread per Column.\n", timeElapse);
	
	// Copy over the result and compare
	hipMemcpy(C_C1, G_C2, size, hipMemcpyDeviceToHost);
	checkResult(C_C, C_C1, N);
	
	// Free all the memory
	hipFree(G_A);
	hipFree(G_B);
	hipFree(G_C);
	hipFree(G_C1);
	hipFree(G_C2);
	free(C_A);
	free(C_B);
	free(C_C);
	free(C_C1);
	hipDeviceReset();
}

int main(){
	computeMatrix(100);
	computeMatrix(200);
	computeMatrix(500);
	computeMatrix(1000);
	computeMatrix(1500);
	computeMatrix(3000);
	computeMatrix(5000);
	printf("------------------------------------------------------------------------\n\n");
    return 0;
}
