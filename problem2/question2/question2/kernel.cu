/*
	Name: Daniyal Manair
	Student Number: 20064993
*/
#include "hip/hip_runtime.h"


#include <vector>
#include <stdio.h>
#include <random>
#include <algorithm>
#include <chrono>
#include <map>

__global__ void sumMatrixGPU(float* A, float* B, float* C, const int N) {
	unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int idx = row * N + col;
	if (row < N && col < N){
		C[idx] = A[idx] + B[idx];
	}
}

__global__ void sumMatrixGPUperRow(float* A, float* B, float* C, const int N) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < N){
		for (int i = 0; i < N; i++)
			C[row * N + i] = A[row * N + i] + B[row * N + i];
	}
}

__global__ void sumMatrixGPUperCol(float* A, float* B, float* C, const int N) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (col < N){
		for (int i = 0; i < N; i++)
			C[i * N + col] = A[i * N + col] + B[i * N + col];
	}
}

void initialData(float* matrix, const int N){
	for (int i = 0; i < (N*N); i++)
		matrix[i] = (float)(rand() & 0xFF) / 10.0f;
}

void sumMatrixCPU(float* A, float* B, float* C, const int N){
	for (int i = 0; i < (N*N); i++)
		C[i] = A[i] + B[i];
}

void checkResult(float* CPU, float* GPU, const int N) {
	double epsilon = 1.0E-8;
	
	for (int i = 0; i < (N*N); i++){
		if (abs(CPU[i] - GPU[i]) > epsilon){
			printf("CPU %f GPU %f ", CPU[i], GPU[i]);
			printf("Arrays do not match.\n\n");
			return;
		}
	}
	printf("Test PASSED\n\n");
}

void printArr(float* matrix, const int N) {
	printf("[");
	for (int i = 0; i < (N*N); i++)
		printf("%f,", matrix[i]);
	printf("\b]\n");

}

void computeMatrix(const int N) {
	// Initial prints
	printf("------------------------------------------------------------------------\n\n");
	printf("%dx%d matrix addition.\n\n", N, N);

	// Initialize Host variables
	float* C_A, *C_B, *C_C, *C_C1;
	float timeDuration;
	size_t size = N * N * sizeof(float);
	hipEvent_t gStart, gEnd;
	FILE *fp;
	
	// Initialize space
	C_A = (float*)malloc(size);
	C_B = (float*)malloc(size);
	C_C = (float*)malloc(size);
	C_C1 = (float*)malloc(size);
	fp=fopen("machineProblem2.csv","a");
	hipEventCreate(&gStart);
    hipEventCreate(&gEnd);
	
	// Set with random data
	initialData(C_A, N);
	initialData(C_B, N);
	memset(C_C, 0, N);
	memset(C_C1, 0, N);

	// Initialize GPU variables
	float* G_A, *G_B, *G_C, *G_C1, *G_C2;
	hipMalloc((void**)&G_A, size);
	hipMalloc((void**)&G_B, size);
	hipMalloc((void**)&G_C, size);
	hipMalloc((void**)&G_C1, size);
	hipMalloc((void**)&G_C2, size);
	
	// Copy over the data
	hipMemcpy(G_A, C_A, size, hipMemcpyHostToDevice);
	hipMemcpy(G_B, C_B, size, hipMemcpyHostToDevice);

	// Serial Test CPU
	auto start = std::chrono::high_resolution_clock::now();
	sumMatrixCPU(C_A, C_B, C_C, N);
	auto end = std::chrono::high_resolution_clock::now();
	auto timeElapse = (std::chrono::duration_cast<std::chrono::microseconds>(end - start).count())/1000.0;
	printf("The CPU took %f to perform the computation.\n\n", timeElapse);
	fprintf(fp,"%d,CPU,ELEMENT,0,%f\n",N,timeElapse);
	
	// Test Complete parallel Computation
	dim3 block(16, 16);
	dim3 thread((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);
	
	hipEventRecord(gStart);
	sumMatrixGPU <<<thread, block >>> (G_A, G_B, G_C, N);
	hipEventRecord(gEnd);
	hipEventSynchronize(gEnd);
	hipEventElapsedTime(&timeDuration, gStart, gEnd);
	printf("The GPU took %f to perform the computation with one thread per element.\n", timeDuration);
	fprintf(fp,"%d,GPU,ELEMENT,16,%f\n",N,timeDuration);
	
	// Copy over the result and compare
	hipMemcpy(C_C1, G_C, size, hipMemcpyDeviceToHost);
	checkResult(C_C, C_C1, N);
	
	// Test row based parallel Computation
	dim3 block1(16);
	dim3 thread1((N + block1.x - 1) / block1.x);
	
	hipEventRecord(gStart);
	sumMatrixGPUperRow <<<thread1, block1 >>> (G_A, G_B, G_C1, N);
	hipEventRecord(gEnd);
	hipEventSynchronize(gEnd);
	hipEventElapsedTime(&timeDuration, gStart, gEnd);
	printf("The GPU took %f to perform the computation with one thread per Row.\n", timeDuration);
	fprintf(fp,"%d,GPU,ROW,16,%f\n",N,timeDuration);

	// Copy over the result and compare
	hipMemcpy(C_C1, G_C1, size, hipMemcpyDeviceToHost);
	checkResult(C_C, C_C1, N);

	// Test Complete parallel Computation
	dim3 block2(16);
	dim3 thread2((N + block2.x - 1) / block2.x);
	
	// Test column based parallel Computation
	hipEventRecord(gStart);
	sumMatrixGPUperCol <<<thread2, block2 >>> (G_A, G_B, G_C2, N);
	hipEventRecord(gEnd);
	hipEventSynchronize(gEnd);
	hipEventElapsedTime(&timeDuration, gStart, gEnd);
	printf("The GPU took %f to perform the computation with one thread per Column.\n", timeDuration);
	fprintf(fp,"%d,GPU,COL,16,%f\n",N,timeDuration);
	
	// Copy over the result and compare
	hipMemcpy(C_C1, G_C2, size, hipMemcpyDeviceToHost);
	checkResult(C_C, C_C1, N);
	
	// Free all the memory
	hipFree(G_A);
	hipFree(G_B);
	hipFree(G_C);
	hipFree(G_C1);
	hipFree(G_C2);
	free(C_A);
	free(C_B);
	free(C_C);
	free(C_C1);
	fclose(fp);
	hipDeviceReset();
}

int main(){
	FILE *fp;
	fp=fopen("machineProblem2.csv","w");
	fprintf(fp,"matrixSize,processor,type,blockSize,time\n");
	fclose(fp);
	computeMatrix(100);
	computeMatrix(200);
	computeMatrix(500);
	computeMatrix(1000);
	computeMatrix(1500);
	computeMatrix(3000);
	computeMatrix(5000);
	printf("------------------------------------------------------------------------\n\n");


    return 0;
}
